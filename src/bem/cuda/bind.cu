#include "hip/hip_runtime.h"
#include "common.h"
#include "gpu_memory.h"
#include "integrand.h"
#include "potential.h"
#include <torch/extension.h>
#include <torch/script.h>
#include <torch/torch.h>
using namespace bem;

torch::Tensor identity_matrix(const torch::Tensor &vertices_, const torch::Tensor &triangles_)
{
    int vertices_size = vertices_.size(0);
    int triangles_size = triangles_.size(0);
    torch::Tensor matrix_ =
        torch::zeros({vertices_size, vertices_size}, torch::dtype(torch::kComplexFloat).device(torch::kCUDA));
    CudaTensor<complex, 2> matrix((complex *)matrix_.data_ptr(), matrix_.size(0), matrix_.size(1));
    float3 *vertices = (float3 *)vertices_.data_ptr();
    int3 *triangles = (int3 *)triangles_.data_ptr();
    parallel_for(triangles_size, [=] __device__(int i) { identityIntegrand(vertices, triangles[i], matrix); });
    return matrix_;
}

torch::Tensor triangle2vertex(const torch::Tensor &vertices_,
                              const torch::Tensor &triangles_,
                              const torch::Tensor neumann_)
{
    int vertices_size = vertices_.size(0);
    int triangles_size = triangles_.size(0);
    torch::Tensor vertex_neumann_ =
        torch::zeros({vertices_size}, torch::dtype(torch::kComplexFloat).device(torch::kCUDA));
    complex *vertex_neumann = (complex *)vertex_neumann_.data_ptr();
    float3 *vertices = (float3 *)vertices_.data_ptr();
    int3 *triangles = (int3 *)triangles_.data_ptr();
    complex *neumann = (complex *)neumann_.data_ptr();
    torch::Tensor vertex_triangle_count_ =
        torch::zeros({vertices_size}, torch::dtype(torch::kInt).device(torch::kCUDA));
    int *vertex_triangle_count = (int *)vertex_triangle_count_.data_ptr();
    parallel_for(triangles_size, [=] __device__(int i) {
        atomicAddCpx(&vertex_neumann[triangles[i].x], neumann[i]);
        atomicAddCpx(&vertex_neumann[triangles[i].y], neumann[i]);
        atomicAddCpx(&vertex_neumann[triangles[i].z], neumann[i]);
        atomicAdd(&vertex_triangle_count[triangles[i].x], 1);
        atomicAdd(&vertex_triangle_count[triangles[i].y], 1);
        atomicAdd(&vertex_triangle_count[triangles[i].z], 1);
    });
    parallel_for(vertices_size, [=] __device__(int i) { vertex_neumann[i] /= vertex_triangle_count[i]; });
    return vertex_neumann_;
}
template <PotentialType type, int LineGaussNum, int TriGaussNum>
torch::Tensor assemble_matrix(const torch::Tensor &vertices_,
                              const torch::Tensor &triangles_,
                              const torch::Tensor &normals_,
                              const torch::Tensor &surface_curls_trans_,
                              const float wave_number)
{
    float3 *vertices = (float3 *)vertices_.data_ptr();
    int3 *triangles = (int3 *)triangles_.data_ptr();
    float3 *normals = (float3 *)normals_.data_ptr();
    int vertices_size = vertices_.size(0);
    int triangles_size = triangles_.size(0);
    torch::Tensor matrix_ =
        torch::zeros({vertices_size, vertices_size}, torch::dtype(torch::kComplexFloat).device(torch::kCUDA));
    CudaTensor<complex, 2> matrix((complex *)matrix_.data_ptr(), matrix_.size(0), matrix_.size(1));
    CudaTensor<float, 3> surface_curls_trans((float *)surface_curls_trans_.data_ptr(), surface_curls_trans_.size(0),
                                             surface_curls_trans_.size(1), surface_curls_trans_.size(2));

    parallel_for_block(triangles_size, 256, [=] __device__(int x, int y) {
        int i = x;
        for (int j = y; j < triangles_size; j += blockDim.x)
        {
            if (triangle_common_vertex_num(triangles[i], triangles[j]) == 0)
            {
                float curl_product[9];
                matmulABT(&surface_curls_trans(i, 0, 0), &surface_curls_trans(j, 0, 0), curl_product);
                face2FaceIntegrandRegular<type, TriGaussNum>(vertices, dot(normals[i], normals[j]), curl_product,
                                                             triangles[i], triangles[j], matrix, wave_number, true);
            }
        }
    });

    parallel_for_block(triangles_size, 256, [=] __device__(int x, int y) {
        int i = x;
        __shared__ int adj[64];
        __shared__ int adj_size;
        if (threadIdx.x == 0)
        {
            adj_size = 0;
        }
        __syncthreads();
        for (int j = y; j < triangles_size; j += blockDim.x)
        {
            int common_vertex_num = triangle_common_vertex_num(triangles[i], triangles[j]);
            if (common_vertex_num > 0)
            {
                adj[atomicAdd_block(&adj_size, 1)] = j;
            }
        }
        __syncthreads();
        const int sub_integrand_size = LineGaussNum * LineGaussNum * LineGaussNum * LineGaussNum;
        __shared__ complex result[64][10];
        __shared__ float normal_prod[64];
        __shared__ float curl_product[64][9];
        for (int j = y; j < adj_size; j += blockDim.x)
        {
            int j_ = adj[j];
            normal_prod[j] = dot(normals[i], normals[j_]);
            matmulABT(&surface_curls_trans(i, 0, 0), &surface_curls_trans(j_, 0, 0), curl_product[j]);
        }
        for (int j = y; j < adj_size * 10; j += blockDim.x)
        {
            result[j / 10][j % 10] = 0;
        }
        __syncthreads();
        for (int j = y; j < adj_size * sub_integrand_size; j += blockDim.x)
        {
            int j_ = j / sub_integrand_size;   // adj index
            int idx = j % sub_integrand_size;  // sub integrand index
            complex local_result[10];
            {
                face2FaceIntegrandSingular<type, LineGaussNum>(vertices, triangles[i], triangles[adj[j_]], wave_number,
                                                               idx, local_result);
                for (int k = 0; k < 10; k++)
                    atomicAddCpxBlock(&result[j_][k], local_result[k]);
            }
        }
        __syncthreads();
        if constexpr (type == HYPER_SINGULAR_LAYER)
        {
            for (int j = y; j < adj_size * 9; j += blockDim.x)
            {
                result[j / 9][j % 9] = curl_product[j / 9][j % 9] * result[j / 9][9] -
                                       result[j / 9][j % 9] * normal_prod[j / 9] * wave_number * wave_number;
            }
            __syncthreads();
        }
        else if constexpr (type == BM_LHS)
        {
            for (int j = y; j < adj_size * 9; j += blockDim.x)
            {
                result[j / 9][j % 9] = curl_product[j / 9][j % 9] * result[j / 9][9] + result[j / 9][j % 9];
            }
            __syncthreads();
        }
        int src_global_idx[3] = {triangles[i].x, triangles[i].y, triangles[i].z};
        for (int j = y; j < adj_size; j += blockDim.x)
        {
            int trg_global_idx[3] = {triangles[adj[j]].x, triangles[adj[j]].y, triangles[adj[j]].z};
            for (int k = 0; k < 3; k++)
                for (int l = 0; l < 3; l++)
                    atomicAddCpx(&matrix(src_global_idx[k], trg_global_idx[l]), result[j][k * 3 + l]);
        }
    });
    return matrix_;
}

template <PotentialType type, int TriGaussNum>
torch::Tensor assemble_potential_matrix(const torch::Tensor &vertices_,
                                        const torch::Tensor &triangles_,
                                        const torch::Tensor &points_,
                                        const float wave_number)
{
    float3 *vertices = (float3 *)vertices_.data_ptr();
    int3 *triangles = (int3 *)triangles_.data_ptr();
    float3 *points = (float3 *)points_.data_ptr();
    int vertices_size = vertices_.size(0);
    int triangles_size = triangles_.size(0);
    int points_size = points_.size(0);
    torch::Tensor matrix_ =
        torch::zeros({points_size, vertices_size}, torch::dtype(torch::kComplexFloat).device(torch::kCUDA));
    CudaTensor<complex, 2> matrix((complex *)matrix_.data_ptr(), matrix_.size(0), matrix_.size(1));
    parallel_for_block(points_size, 512, [=] __device__(int i, int j) {
        for (int k = j; k < triangles_size; k += blockDim.x)
            face2PointIntegrand<type, TriGaussNum>(vertices, triangles[k], points[i], wave_number, &matrix(i, 0));
    });
    return matrix_;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("single_boundary_matrix", &assemble_matrix<bem::SINGLE_LAYER, 4, 6>,
          "Assemble single layer matrix with 4 and 6 gauss points");
    m.def("single_boundary_matrix_approx", &assemble_matrix<bem::SINGLE_LAYER, 2, 3>,
          "Assemble single layer matrix with 2 and 3 gauss points");
    m.def("single_boundary_matrix_approx_1", &assemble_matrix<bem::SINGLE_LAYER, 2, 1>,
          "Assemble single layer matrix with 1 gauss points");
    m.def("double_boundary_matrix", &assemble_matrix<bem::DOUBLE_LAYER, 4, 6>,
          "Assemble double layer matrix with 4 and 6 gauss points");
    m.def("double_boundary_matrix_approx", &assemble_matrix<bem::DOUBLE_LAYER, 2, 3>,
          "Assemble double layer matrix with 2 and 3 gauss points");
    m.def("double_boundary_matrix_approx_1", &assemble_matrix<bem::DOUBLE_LAYER, 2, 1>,
          "Assemble double layer matrix with 1 gauss points");
    m.def("hypersingular_boundary_matrix", &assemble_matrix<bem::HYPER_SINGULAR_LAYER, 4, 6>,
          "Assemble hypersingular layer matrix with 4 and 6 gauss points");
    m.def("hypersingular_boundary_matrix_approx", &assemble_matrix<bem::HYPER_SINGULAR_LAYER, 2, 3>,
          "Assemble hypersingular layer matrix with 2 and 3 gauss points");
    m.def("hypersingular_boundary_matrix_approx_1", &assemble_matrix<bem::HYPER_SINGULAR_LAYER, 2, 1>,
          "Assemble hypersingular layer matrix with 1 gauss points");
    m.def("adjointdouble_boundary_matrix", &assemble_matrix<bem::ADJOINT_DOUBLE_LAYER, 4, 6>,
          "Assemble adjoint double layer matrix with 4 and 6 gauss points");
    m.def("adjointdouble_boundary_matrix_approx", &assemble_matrix<bem::ADJOINT_DOUBLE_LAYER, 2, 3>,
          "Assemble adjoint double layer matrix with 2 and 3 gauss points");
    m.def("adjointdouble_boundary_matrix_approx_1", &assemble_matrix<bem::ADJOINT_DOUBLE_LAYER, 2, 1>,
          "Assemble adjoint double layer matrix with 1 gauss points");
    m.def("bm_lhs_boundary_matrix", &assemble_matrix<bem::BM_LHS, 4, 6>,
          "Assemble BM LHS matrix with 4 and 6 gauss points");
    m.def("bm_lhs_boundary_matrix_approx", &assemble_matrix<bem::BM_LHS, 2, 3>,
          "Assemble BM LHS matrix with 2 and 3 gauss points");
    m.def("bm_lhs_boundary_matrix_approx_1", &assemble_matrix<bem::BM_LHS, 2, 1>,
          "Assemble BM LHS matrix with 1 gauss points");
    m.def("bm_rhs_boundary_matrix", &assemble_matrix<bem::BM_RHS, 4, 6>,
          "Assemble BM RHS matrix with 4 and 6 gauss points");
    m.def("bm_rhs_boundary_matrix_approx", &assemble_matrix<bem::BM_RHS, 2, 3>,
          "Assemble BM RHS matrix with 2 and 3 gauss points");
    m.def("bm_rhs_boundary_matrix_approx_1", &assemble_matrix<bem::BM_RHS, 2, 1>,
          "Assemble BM RHS matrix with 1 gauss points");
    m.def("identity_matrix", &identity_matrix, "Assemble identity matrix");
    m.def("triangle2vertex", &triangle2vertex, "Assemble triangle to vertex");

    m.def("single_boundary_potential", &assemble_potential_matrix<bem::SINGLE_LAYER, 6>,
          "Assemble single layer potential matrix with 6 gauss points");
    m.def("double_boundary_potential", &assemble_potential_matrix<bem::DOUBLE_LAYER, 6>,
          "Assemble double layer potential matrix with 6 gauss points");
}
